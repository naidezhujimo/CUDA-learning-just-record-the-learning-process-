#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define M 3
#define K 4
#define N 2

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

#define PRINT_MATRIX(mat, rows, cols) \
    for (int i = 0; i < rows; i++) { \
        for (int j = 0; j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void CPU_matmul(float *A, float *B, float *C) {
    for (int i=0; i<M; i++) {
        for (int j=0; j<K; j++){
            float sum = 0.0f;
            for (int k=0; k<N; k++){
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

int main() {
    float A[M * K] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f};
    float B[K * N] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f};
    float C_cpu[M * N], C_cublas_s[M * N], C_cublas_h[M * N];

    CPU_matmul(A, B, C_cpu);
    
    // 创建cublas句柄
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // 调用cuBLAS的 cublasSgemm 函数，执行单精度浮点矩阵乘法 C=α×B×A+β×C
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
    CHECK_CUDA(hipMemcpy(C_cublas_s, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    half *d_A_h, *d_B_h, *d_C_h;
    CHECK_CUDA(hipMalloc(&d_A_h, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B_h, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C_h, M * N * sizeof(half)));

    // 创建半精度浮点数矩阵
    half A_h[M * K], B_h[K * N];
    for (int i = 0; i < M * K; i++){
        A_h[i] = __float2half(A[i]);
    }
    for (int i = 0; i < K * N; i++){
        B_h[i] = __float2half(B[i]);
    }

    CHECK_CUDA(hipMemcpy(d_A_h, A_h, M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_h, B_h, K * N * sizeof(half), hipMemcpyHostToDevice));

    // 调用cuBLAS的 cublasHgemm 函数，执行半精度浮点矩阵乘法 C=α×B×A+β×C。
    __half alpha_h = __float2half(1.0f), beta_h = __float2half(0.0f);
    CHECK_CUBLAS(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha_h, d_B_h, N, d_A_h, K, &beta_h, d_C_h, N));

    half C_h[M * N];
    CHECK_CUDA(hipMemcpy(C_h, d_C_h, M * N * sizeof(half), hipMemcpyDeviceToHost));
    // 将半精度浮点结果转换为单精度浮点
    for (int i = 0; i < M * N; i++) {
        C_cublas_h[i] = __half2float(C_h[i]);
    }

    printf("Matrix A (%dx%d):\n", M, K);
    PRINT_MATRIX(A, M, K);
    printf("Matrix B (%dx%d):\n", K, N);
    PRINT_MATRIX(B, K, N);
    printf("CPU Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cpu, M, N);
    printf("cuBLAS SGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_s, M, N);
    printf("cuBLAS HGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_h, M, N);

    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_A_h));
    CHECK_CUDA(hipFree(d_B_h));
    CHECK_CUDA(hipFree(d_C_h));
    CHECK_CUBLAS(hipblasDestroy(handle));
}