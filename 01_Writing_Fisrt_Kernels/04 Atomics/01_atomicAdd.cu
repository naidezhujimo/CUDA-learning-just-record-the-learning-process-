#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 1000
#define NUM_BLOCKS 1000

__global__ void incrementCounterNonAtomic(int* counter) {
    int old = *counter;
    int new_value = old + 1;
    *counter = new_value;
}

__global__ void incrementCounterAtomic(int* counter) {
    int a = atomicAdd(counter, 1);
}

int main() {
    int h_counterNonAtomic = 0;
    int h_counterAtomic = 0;
    int *d_counterNonAtomic, *d_counterAtomic;

    hipMalloc((void**)&d_counterNonAtomic, sizeof(int));
    hipMalloc((void**)&d_counterAtomic, sizeof(int));

    hipMemcpy(d_counterNonAtomic, &h_counterNonAtomic, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_counterAtomic, &h_counterAtomic, sizeof(int), hipMemcpyHostToDevice);

    incrementCounterNonAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterNonAtomic);
    incrementCounterAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterAtomic);

    hipMemcpy(&h_counterNonAtomic, d_counterNonAtomic, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_counterAtomic, d_counterAtomic, sizeof(int), hipMemcpyDeviceToHost);

    printf("Non-atomic counter value: %d\n", h_counterNonAtomic);
    printf("Atomic counter value: %d\n", h_counterAtomic);

    hipFree(d_counterNonAtomic);
    hipFree(d_counterAtomic);

    return 0;
}