#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <windows.h>

#define M 256
#define K 512
#define N 256
#define BLOCK_SIZE 32

// 矩阵A为M×K、矩阵B为K×N、输出矩阵C为M×N

// CPU矩阵乘法
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int l = 0; l < k; l++) {
                sum += A[i * k + l] * B[l * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

// GPU矩阵乘法
__global__ void matmul_gpu(float *A, float *B, float *C, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float sum = 0.0f;
        for (int l = 0; l < k; l++) {
            sum += A[row * k + l] * B[l * n + col];
        }
        C[row * n + col] = sum;
    }
}

void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand / RAND_MAX;
    }
}


double get_time() {
    LARGE_INTEGER frequency;
    LARGE_INTEGER time;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&time);
    return (double)time.QuadPart / frequency.QuadPart;
}


int main() {
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
    }

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;


    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;


    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);


    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}